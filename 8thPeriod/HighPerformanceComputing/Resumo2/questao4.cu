#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void soma(int *vetorA, int *vetorB, int *vetorC, int tam, int iter) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx < tam) {
    vetorC[idx] = vetorA[idx] + vetorB[idx];
    printf("Iter=%d, Blc=%d, Thread=%d, Tam=%d, VetC[%d]=%d \n", iter,
           blockIdx.x, threadIdx.x, tam, idx, vetorC[idx]);
  }
}
int main(int argc, char **argv) {
  int i, *vetorA, *vetorB, *vetorC, threadsPerBlock;
  int *vetorA_d, *vetorB_d, *vetorC_d;

  hipStream_t stream;

  hipStreamCreate(&stream);

  int tam = 16;

  threadsPerBlock = 2;

  int streams_nr = 2;

  int threadsPerGrid = tam / streams_nr;

  int blocksPerGrid = (threadsPerGrid + threadsPerBlock - 1) / threadsPerBlock;

  hipHostMalloc((void **)&vetorA, tam * (sizeof(int)));
  hipHostMalloc((void **)&vetorB, tam * (sizeof(int)));
  hipHostMalloc((void **)&vetorC, tam * (sizeof(int)));
  hipMalloc((void **)&vetorA_d, threadsPerGrid * (sizeof(int)));
  hipMalloc((void **)&vetorB_d, threadsPerGrid * (sizeof(int)));
  hipMalloc((void **)&vetorC_d, threadsPerGrid * (sizeof(int)));
  for (i = 0; i < tam; i++) {
    vetorA[i] = i;
    vetorB[i] = 0;
  }
  printf("Host: tam=%d, streams_nr=%d, TPG=%d, BPG=%d, TPB=%d \n", tam,
         streams_nr, threadsPerGrid, blocksPerGrid, threadsPerBlock);
  for (i = 0; i < tam; i += threadsPerGrid) {
    if ((tam - i) < threadsPerGrid)
      threadsPerGrid = tam - i;
    hipMemcpyAsync(vetorA_d, vetorA + i, threadsPerGrid * (sizeof(int)),
                    hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(vetorB_d, vetorB + i, threadsPerGrid * (sizeof(int)),
                    hipMemcpyHostToDevice, stream);
    soma<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
        vetorA_d, vetorB_d, vetorC_d, threadsPerGrid, i);

    hipMemcpyAsync(vetorC + i, vetorC_d, threadsPerGrid * (sizeof(int)),
                    cu daMemcpyDeviceToHost, stream);
  }
  hipStreamSynchronize(stream);
  for (i = 0; i < tam; i++) {
    printf("%d ", vetorC[i]);
  }
  hipHostFree(vetorA);
  hipHostFree(vetorB);
  hipHostFree(vetorC);
  hipFree(vetorA_d);
  hipFree(vetorB_d);

  hipFree(vetorC_d);
  hipStreamDestroy(stream);
}