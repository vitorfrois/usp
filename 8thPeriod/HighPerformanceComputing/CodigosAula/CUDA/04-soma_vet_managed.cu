/*
Soma dois vetores
Ilustra o uso da memória unificada (managed memory)
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define TAM 16
#define TPB 256

__global__ void soma(int *vetA_glb, int *vetB_glb,int *vetC_glb){ 
    int idx = blockDim.x * blockIdx.x + threadIdx.x; 

    if (idx < TAM) 
    {
        vetC_glb[idx] = vetA_glb[idx] + vetB_glb[idx];
    }
}

int main(int argc,char **argv){
	int *vetA,*vetB,*vetC;
	int blocksPerGrid; 

    int i;
    
    //Aloca os vetores no host e no device
    hipMallocManaged((void**)&vetA,TAM*(sizeof(int))); 
    hipMallocManaged((void**)&vetB,TAM*(sizeof(int))); 
    hipMallocManaged((void**)&vetC,TAM*(sizeof(int))); 
    
    //Preenche os vetores no host
    for(i=0;i<TAM;i++){
        vetA[i]=i;
        vetB[i]=0; //-i;
    }

    //Define a quantidade de blocos por grade
    blocksPerGrid=(TAM+TPB-1)/TPB; 
    
    //Calcula o tempo de execução do kernel com eventos
    hipEvent_t start, stop;
    float gpu_time = 0.0f;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    //Invoca o kernel com blocksPerGrid blocos e threadsPerBlock threads
    soma <<<blocksPerGrid,TPB>>> (vetA,vetB,vetC); 
    
    //Calcula o tempo de execução do kernel com eventos
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    //Sincroniza as threads do device para impressão do resultado
    hipDeviceSynchronize();

    //Imprime o resultado no host
    for(i=0;i<TAM;i++){
       printf("%d ",vetC[i]);
    }

    printf("\nElapsed Time:%.5f (ms)\n", gpu_time);
    
    //Desaloca os vetores no device
    hipFree(vetA);
    hipFree(vetB);
    hipFree(vetC);
}