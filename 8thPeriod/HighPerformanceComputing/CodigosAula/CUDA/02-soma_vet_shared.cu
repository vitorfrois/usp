/*
Soma dois vetores
Ilustra a alocação dinâmica da memoria compartilhada
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define TAM 16
#define TPB 256

__global__ void soma(int *vetA_glb, int *vetB_glb,int *vetC_glb){ 

	// alocacao dinamica de vetC_shd
    extern __shared__ int vetC_shd[];

    int idx = blockDim.x * blockIdx.x + threadIdx.x; 

    if (idx < TAM) 
    {
        vetC_shd[idx] = vetA_glb[idx]+vetB_glb[idx];
        vetC_glb[idx] = vetC_shd[idx];
    }
}

int main(int argc,char **argv){
    int *vetA_h,*vetB_h,*vetC_h;
    int *vetA_d,*vetB_d,*vetC_d;

	int i, blocksPerGrid; 

    //Aloca os vetores no host
    vetA_h=(int *)malloc(TAM * sizeof(int)); 
    vetB_h=(int *)malloc(TAM * sizeof(int)); 
    vetC_h=(int *)malloc(TAM * sizeof(int)); 

    //Aloca os vetores no device
    hipMalloc((void**)&vetA_d,TAM*(sizeof(int))); 
    hipMalloc((void**)&vetB_d,TAM*(sizeof(int))); 
    hipMalloc((void**)&vetC_d,TAM*(sizeof(int))); 
    
    //Preenche os vetores no host
    for(i=0;i<TAM;i++){
        vetA_h[i]=i;
        vetB_h[i]=0; //-i;
    }
    
    //Copia o conteúdo dos vetores para o device
    hipMemcpy(vetA_d,vetA_h,TAM*(sizeof(int)), hipMemcpyHostToDevice); 
    hipMemcpy(vetB_d,vetB_h,TAM*(sizeof(int)), hipMemcpyHostToDevice); 

    //Define a quantidade de blocos por grade
    blocksPerGrid=(TAM+TPB-1)/TPB; 
	
    //Calcula o tempo de execução do kernel com eventos
    // cudaEvent_t start, stop;
    // float gpu_time = 0.0f;
    // cudaEventCreate(&start);
    // cudaEventCreate(&stop);
    // cudaEventRecord(start, 0);
    
    //Invoca o kernel com blocksPerGrid blocos e threadsPerBlock threads
    soma <<<blocksPerGrid,TPB,TAM*sizeof(int)>>> (vetA_d,vetB_d,vetC_d);
    
    //Calcula o tempo de execução do kernel com eventos
    // cudaDeviceSynchronize();
    // cudaEventRecord(stop, 0);
    // cudaEventSynchronize(stop);
    // cudaEventElapsedTime(&gpu_time, start, stop);
    // printf("%.5f\n", gpu_time);
    // cudaEventDestroy(start);
    // cudaEventDestroy(stop);

    //Copia o resultado da soma de volta para o host
    hipMemcpy(vetC_h,vetC_d,TAM*(sizeof(int)), hipMemcpyDeviceToHost);

    //Imprime o resultado no host
    for(i=0;i<TAM;i++){
       printf("%d ",vetC_h[i]);
    }
    
    //Desaloca os vetores no host
    free(vetA_h); 
    free(vetB_h);
    free(vetC_h);  
    
    //Desaloca os vetores no device
    hipFree(vetA_d);
    hipFree(vetB_d);
    hipFree(vetC_d);
}