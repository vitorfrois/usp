/*
Soma duas matrizes quadradas 
Ilustra o uso da mem global e mem local

*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define TAM 16
#define THREADS 3

__global__ void soma(int *A_dev_glb, int *B_dev_glb,int *C_dev_glb)
{ 
    int i = blockDim.x * blockIdx.x + threadIdx.x; 
    int j = blockDim.y * blockIdx.y + threadIdx.y; 

    if (i < TAM && j < TAM) 
    {
        C_dev_glb[i*TAM+j] = A_dev_glb[i*TAM+j] + B_dev_glb[i*TAM+j];
    }
}

int main(int argc,char **argv)
{
	int *mA_hst,*mB_hst,*mC_hst; 
    int *mA_dev,*mB_dev,*mC_dev;
    int i,j;
    
    //Aloca matrizes no host
    mA_hst=(int *)malloc(TAM*TAM*sizeof(int)); 
    mB_hst=(int *)malloc(TAM*TAM*sizeof(int)); 
    mC_hst=(int *)malloc(TAM*TAM*sizeof(int)); 

    //Aloca matrizes no device
    hipMalloc((void**)&mA_dev,TAM*TAM*(sizeof(int))); 
    hipMalloc((void**)&mB_dev,TAM*TAM*(sizeof(int))); 
    hipMalloc((void**)&mC_dev,TAM*TAM*(sizeof(int))); 

    //Preenche matrizes no host
    for(i=0;i<TAM;i++)
	{
        for(j=0;j<TAM;j++)
		{
            mA_hst[i*TAM+j]=i;
            mB_hst[i*TAM+j]=-i;
        }
    }

    //Copia o conteúdo das matrizes para o device
    hipMemcpy(mA_dev,mA_hst,TAM*TAM*(sizeof(int)), hipMemcpyHostToDevice); 
    hipMemcpy(mB_dev,mB_hst,TAM*TAM*(sizeof(int)), hipMemcpyHostToDevice); 

    //Define a quantidade de threads por bloco
    dim3 threadsPerBlock(THREADS,THREADS);

    //Define a quantidade de blocos por grade
    dim3 blocksPerGrid((TAM+(threadsPerBlock.x-1)) / threadsPerBlock.x, (TAM+(threadsPerBlock.y-1)) / threadsPerBlock.y);

    //Invoca o kernel com blocksPerGrid blocos e threadsPerBlock threads
    soma <<<blocksPerGrid,threadsPerBlock>>> (mA_dev,mB_dev,mC_dev); 

    //Copia o resultado da soma de volta para o host
    hipMemcpy(mC_hst,mC_dev,TAM*TAM*(sizeof(int)), hipMemcpyDeviceToHost);

    //Imprime o resultado no host. Matriz C deve ter apenas valores zero.
    for(i=0;i<TAM;i++)
	{
        for(j=0;j<TAM;j++)
            printf("%d ",mC_hst[i*TAM+j]);
        printf("\n");
    }
    
    //Desaloca matrizes no host
    free(mA_hst); 
    free(mB_hst);
    free(mC_hst);  
    
    //Desaloca matrizes no device
    hipFree(mA_dev);
    hipFree(mB_dev);
    hipFree(mC_dev);
}
