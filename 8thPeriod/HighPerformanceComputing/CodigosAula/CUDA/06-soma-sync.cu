// codigo incrementa e depois decrementa valores de um vetor.
//
// este codigo exemplifica o uso de __syncthreads() e 
// o uso de memoria compartilhada criada estaticamente
// e dinamicamente.
//
// a primeira grade incrementa as posicoes de um vetor
// N vezes por thread.  Usa memoria compartilhada criada estaticamente
//
// a segunda grade decrementa da mesma forma.
// o vetor comeca com vlrs 0 (zero) e termina com 0 (zero).
// Usa memória compartilhada criada dinamicamente.
//

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// no maximo 1024, pois determina o nr de threads no bloco
#define N 1024

__global__ void staticShdMem(int *vetD_glb) 
{
  // criacao estatica na memoria compartilhada
  __shared__ int vetD_shd[N];
  int t = threadIdx.x;

  int i, aux;

  // transfere dado da memoria global para a mem shd
  // as N threads fazem isso
  vetD_shd[t] = vetD_glb[t];
  
  // cada thread incrementa N vezes cada posicao t
  for (i = 0; i < N; i++)
  {
	__syncthreads(); // sync necessaria
	aux = vetD_shd[N-t-1];
	aux++;
	__syncthreads(); // apenas esta sync produz resultados errados
	vetD_shd[t] = aux;
  }

  // Retorna resultado para memoria global. 
  // Executado por todas as N threads.
  vetD_glb[t] = vetD_shd[t];
  
}

__global__ void dynamicShdMem(int *vetD_glb)
{
  // criacao dinamica na memoria compartilhada
  extern __shared__ int vetD_shd[];

  int t = threadIdx.x;
  
  int i, aux;

  // transfere dado da memoria global para a mem shd
  // as N threads fazem isso
  vetD_shd[t] = vetD_glb[t];  
  
  // cada thread decrementa N vezes cada posicao t
  for (i = 0; i < N; i++)
  {
	__syncthreads(); // sync necessaria
	aux = vetD_shd[N-t-1];
	aux--;
	__syncthreads(); // apenas esta sync produz resultados errados
	vetD_shd[t] = aux;
  }

  // Retorna resultado para memoria global. 
  // Executado por todas as N threads.
  vetD_glb[t] = vetD_shd[t];
}


int main(void)
{
  int vetA_h[N], vetD_h[N];
  int *vetD_d;
  int i;

  for (i = 0; i < N; i++) {
    vetA_h[i] = 0;
  }

  hipMalloc(&vetD_d, N * sizeof(int)); 

  hipMemcpy(vetD_d, vetA_h, N*sizeof(int), hipMemcpyHostToDevice);
  
  // algoritmo so funciona se as threads estiverem no mesmo bloco
  // devido a limitacao do __syncthreads()
  // invoca a versao com memoria compartilhada criada estaticamente
  staticShdMem<<<1,N>>>(vetD_d);
  
  hipMemcpy(vetD_h, vetD_d, N*sizeof(int), hipMemcpyDeviceToHost);

  printf("Static Results(%d): ", N);
  for (i = 0; i < N; i++) 
		printf("%d ", vetD_h[i]);
  printf("\n\n");
		

//  nao eh necessario copiar o vetor d do host para o device. 
//  a grade anterior fez essa copia e o conteudo dele ainda esta no device
//  cudaMemcpy(vetD_d, vetD_h, N*sizeof(int), cudaMemcpyHostToDevice);

  // invoca a versao com memoria compartilhada criada dinamicamente
  dynamicShdMem<<<1,N,(N*sizeof(int))>>>(vetD_d);
  
  hipMemcpy(vetD_h, vetD_d, N * sizeof(int), hipMemcpyDeviceToHost);

  printf("Dynamic Results(%d): ", N);
  for (i = 0; i < N; i++) 
		printf("%d ", vetD_h[i]);
  printf("\n\n");

  //Desaloca vetor no device
  hipFree(vetD_d);

  exit(0);
}
