/*
Multiplica um vetor por uma constante.
Exemplo para o uso de memória constante em CUDA
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define TAM 100
#define VLR_ESCALAR 10
#define TPB 256

__device__ __constant__ int escalar_d;

__global__ void mult(int *vetA_glb){ 

    int idx = blockDim.x * blockIdx.x + threadIdx.x; 

    if (idx < TAM) 
    {
        vetA_glb[idx] = escalar_d * vetA_glb[idx];
    }
}

int main(int argc,char **argv){
    int *vetA_h;
    int *vetA_d;

	int blocksPerGrid;
	
	int i, escalar_h; 


    //Aloca o vetor no host
    vetA_h=(int *)malloc(TAM * sizeof(int)); 

    //Aloca o vetor no device
    hipMalloc((void**)&vetA_d,TAM*(sizeof(int))); 
    
    //Preenche o vetor no host
    for(i=0;i<TAM;i++){
        vetA_h[i]=i;
    }

    //Copia o conteúdo do vetor para o device
    hipMemcpy(vetA_d,vetA_h,TAM*(sizeof(int)), hipMemcpyHostToDevice); 

    escalar_h=VLR_ESCALAR;
    //Copia o conteúdo de escalar_h, lido do terminal, para a variável constante escalar_d, no device
    hipMemcpyToSymbol(HIP_SYMBOL(escalar_d),&escalar_h,sizeof(int));

    //Define a quantidade de blocos por grade
    blocksPerGrid=(TAM+TPB-1)/TPB; 

    //Invoca o kernel com blocksPerGrid blocos e threadsPerBlock threads
    mult <<<blocksPerGrid,TPB>>> (vetA_d); 

    //Copia o resultado da soma de volta para o host
    hipMemcpy(vetA_h,vetA_d,TAM*(sizeof(int)), hipMemcpyDeviceToHost);

    //Imprime o resultado no host
    for(i=0;i<TAM;i++){
       printf("%d ",vetA_h[i]);
    }
    
    //Desaloca os vetores no host
    free(vetA_h); 
    
    //Desaloca os vetores no device
    hipFree(vetA_d);
}