/*
Troca os valores de posição em um vetor (inverte os valores no vetor).

Exemplo da necessidade da sincronização de threads de um bloco.
Exemplo para alocação dinâmica e estática de shared mem

Quando a função __syncthreads() no kernel está comentada, o resultado fica errado.
Os if's nos for's das saídas dos resultados mostram os casos errados.
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 1024

__global__ void staticReverse(int *vetD_glb)
{
  __shared__ int vetD_shd[N];
  int t = threadIdx.x;
  int tr = N-t-1;

  vetD_shd[t] = vetD_glb[t];
  __syncthreads();
  vetD_glb[t] = vetD_shd[tr];
}

__global__ void dynamicReverse(int *vetD_glb)
{
  extern __shared__ int vetD_shd[];
  int t = threadIdx.x;
  int tr = N-t-1;

  vetD_shd[t] = vetD_glb[t];
  __syncthreads();
  vetD_glb[t] = vetD_shd[tr];
}

int main(void)
{
  int vetA_h[N], vetCtrl_h[N], vetD_h[N];
  int *vetD_d;
  int i;

  for (i = 0; i < N; i++) {
    vetA_h[i] = i;  // source
    vetCtrl_h[i] = N-i-1;  // just to check our results
    vetD_h[i] = 0;   // target
  }

  hipMalloc(&vetD_d, N * sizeof(int)); 

  // copy vet a into device global memory
  hipMemcpy(vetD_d, vetA_h, N*sizeof(int), hipMemcpyHostToDevice);

  // run version with static shared memory
  staticReverse<<<1,N>>>(vetD_d);

  // copy results from device to host memory
  hipMemcpy(vetD_h, vetD_d, N*sizeof(int), hipMemcpyDeviceToHost);

  printf("Static Results(%d): ", N);
  for (i = 0; i < N; i++) 
	if (vetD_h[i] != vetCtrl_h[i])
		printf("vetD_h[%d]=%d, vetCtrl_h[%d]=%d \n", i, vetD_h[i], i, vetCtrl_h[i]);

  printf("\nN=%d \n", N);

  // *************************************************
  
    // copy again vet a into device global memory
  hipMemcpy(vetD_d, vetA_h, N*sizeof(int), hipMemcpyHostToDevice);
  
  // run dynamic shared memory version
  dynamicReverse<<<1,N,N*sizeof(int)>>>(vetD_d);

  // copy results from device to host memory
  hipMemcpy(vetD_h, vetD_d, N * sizeof(int), hipMemcpyDeviceToHost);

  printf("Dynamic Results(%d): ", N);
  for (i = 0; i < N; i++) 
	if (vetD_h[i] != vetCtrl_h[i])
		printf("vetD_h[%d]=%d, vetCtrl_h[%d]=%d \n", i, vetD_h[i], i, vetCtrl_h[i]);
  printf("\nN=%d \n", N);

  
  // device memory free!!!!
  hipFree(vetD_d);

  exit(0);
}
